﻿// CUDAPlayground
// 
// Tutorial
//	Programming Guide https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
//  Warp Functions https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#warp-vote-functions
//
// Sample
//	CUDA Samples https://github.com/NVIDIA/cuda-samples
//	Optimizing Parallel Reduction in CUDA https://cuvilib.com/Reduction.pdf
//


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void cuda_hello() 
{
	// Grid -> Block -> Thread
	int block_size = blockDim.x * blockDim.y * blockDim.z;
	int tid = 
		blockIdx.z * (gridDim.x * gridDim.y) * block_size +
		blockIdx.y * (gridDim.x) * block_size +
		blockIdx.x * block_size +
		threadIdx.z * (blockDim.x * blockDim.y) +
		threadIdx.y * (blockDim.x) + 
		threadIdx.x;
	printf("tid = %2d; blockIdx = %d,%d,%d / %d,%d,%d; threadIdx = %d,%d,%d / %d,%d,%d\n", 
		tid,
		blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z,
		threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
}

int main()
{
	dim3 grid_size(1, 1, 1);
	dim3 block_size(8, 4, 1);
	cuda_hello<<<grid_size, block_size>>>();

	return 0;
}
