﻿// CUDAPlayground
// 
// Tutorial
//	Programming Guide https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
//  Warp Functions https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#warp-vote-functions
//
// Sample
//	CUDA Samples https://github.com/NVIDIA/cuda-samples
//	Optimizing Parallel Reduction in CUDA https://cuvilib.com/Reduction.pdf
//

#include <hip/hip_runtime.h>						// Main CUDA runtime API
			// Required for __launch_bounds__, blockDim, gridDim, etc.

#include <stdio.h>

__global__ void cuda_hello_world() 
{
	// Grid -> Block -> Thread
	int block_size = blockDim.x * blockDim.y * blockDim.z;
	int tid = 
		blockIdx.z * (gridDim.x * gridDim.y) * block_size +
		blockIdx.y * (gridDim.x) * block_size +
		blockIdx.x * block_size +
		threadIdx.z * (blockDim.x * blockDim.y) +
		threadIdx.y * (blockDim.x) + 
		threadIdx.x;
	printf("tid = %2d; blockIdx = %d,%d,%d / %d,%d,%d; threadIdx = %d,%d,%d / %d,%d,%d\n", 
		tid,
		blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z,
		threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
}

int main()
{
	dim3 grid_size(1, 1, 1);					// Dispatch in HLSL
	dim3 block_size(8, 4, 1);					// ThreadGroup in HLSL 
	cuda_hello_world<<<grid_size, block_size>>>();

	return 0;
}
